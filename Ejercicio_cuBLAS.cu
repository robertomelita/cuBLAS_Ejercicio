#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#define IDX2C(i,j,ld) (((j)*(ld))+(i)) 

#define m 6 // a - mxk matrix
#define n 4 // b - kxn matrix
#define k 5 // c - mxn matrix

__host__ void printMatrix(float *c, int m_, int n_);
__host__ void initMatrix(float *c, int m_, int n_, int ind);

int main(void){
    hipError_t cudaStat ; // cudaMalloc status
    hipblasStatus_t stat ; // CUBLAS functions status
    hipblasHandle_t handle ; // CUBLAS context

    float *a; // m x k matrix a on the host
    float *b; // k x n matrix b on the host
    float *c; // m x n matrix c on the host

    float al =  1.0f; // alpha
    float bet = 0.0f; // beta

    a =(float*)malloc(m*k*sizeof(float)); // host memory for a
    b =(float*)malloc(k*n*sizeof(float)); // host memory for b
    c =(float*)malloc(m*n*sizeof(float)); // host memory for c

    // define an mxk matrix a
    initMatrix(a,m,k,11);
    printMatrix(a,m,k);

    // define a kxn matrix b 
    initMatrix(b,k,n,11);
    printMatrix(b,k,n);

    // define an mxn matrix c 
    initMatrix(c,m,n,11);    
    printMatrix(c,m,n);



    

    // 1) Crear Matrices en GPU y reservarles memoria
    //


    // 2) Inicializar cuBLAS
    //


    // 3) Copiar matrices desde host a device (RAM -> VRAM)
    //


    // 4) Realizar la multiplicacion de matrices
    //
    // AYUDA:
    /*

    cublasStatus_t cublasSgemm(cublasHandle_t handle,
                            cublasOperation_t transa, 
                            cublasOperation_t transb,
                            int m, 
                            int n, 
                            int k,
                            const float *alpha,
                            const float *A, int lda,
                            const float *B, int ldb,
                            const float *beta,
                            float *C, int ldc)
    */

    // 5) Copiar matriz resultado desde device a host (VRAM -> RAM)
    //

    // 6) Destruir CUBLAS context
    //





    // print result
    printf("Result:\n");
    printMatrix(c,m,n);

    free(a); // free host memory
    free(b); // free host memory
    free(c); // free host memory
    
    return EXIT_SUCCESS;
}


__host__ void printMatrix(float *c, int m_, int n_){
    for(int i = 0; i < m_; i++){
        for(int j = 0;j < n_; j++){
            printf(" %7.0f",c[IDX2C(i,j,m_)]); // print c after Sgemm
        }
        printf("\n");
    }
    printf("\n");
}
__host__ void initMatrix(float *c, int m_, int n_, int ind){
    for(int i = 0; i < m_; i++){    
        for(int j = 0; j < n_; j++){ 
            c[IDX2C(i,j,m_)] =(float)ind++;
        }    
    }
}

